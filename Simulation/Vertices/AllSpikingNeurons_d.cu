/*
 * AllSpikingNeurons_d.cu
 *
 */

#include "AllSpikingNeurons.h"
#include "AllSpikingSynapses.h"
#include "Book.h"

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllSpikingNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingNeurons::copyDeviceSpikeHistoryToHost( AllSpikingNeuronsDeviceProperties& allNeurons ) 
{
        int numNeurons = Simulator::getInstance().getTotalNeurons();
        uint64_t* pSpikeHistory[numNeurons];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spikeHistory_, numNeurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );

        int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
        for (int i = 0; i < numNeurons; i++) {
                HANDLE_ERROR( hipMemcpy ( spikeHistory_[i], pSpikeHistory[i],
                        maxSpikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllSpikingNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingNeurons::copyDeviceSpikeCountsToHost( AllSpikingNeuronsDeviceProperties& allNeurons ) 
{
        int numNeurons = Simulator::getInstance().getTotalNeurons();

        HANDLE_ERROR( hipMemcpy ( spikeCount_, allNeurons.spikeCount_, numNeurons * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( spikeCountOffset_, allNeurons.spikeCountOffset_, numNeurons * sizeof( int ), hipMemcpyDeviceToHost ) );
}

/*
 *  Clear the spike counts out of all neurons in device memory.
 *  (helper function of clearNeuronSpikeCounts)
 *
 *  @param  allNeurons         Reference to the allNeurons struct.
 */
void AllSpikingNeurons::clearDeviceSpikeCounts( AllSpikingNeuronsDeviceProperties& allNeurons ) 
{
        int numNeurons = Simulator::getInstance().getTotalNeurons();

        HANDLE_ERROR( hipMemset( allNeurons.spikeCount_, 0, numNeurons * sizeof( int ) ) );
        HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCountOffset_, spikeCountOffset_, numNeurons * sizeof( int ), hipMemcpyHostToDevice ) );
}

/*
 *  Set some parameters used for advanceNeuronsDevice.
 *  Currently we set the two member variables: m_fpPreSpikeHit_h and m_fpPostSpikeHit_h.
 *  These are function pointers for PreSpikeHit and PostSpikeHit device functions
 *  respectively, and these functions are called from advanceNeuronsDevice device
 *  function. We use this scheme because we cannot not use virtual function (Polymorphism)
 *  in device functions.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 */
void AllSpikingNeurons::setAdvanceNeuronsDeviceParams(IAllSynapses &synapses)
{
    AllSpikingSynapses &spSynapses = dynamic_cast<AllSpikingSynapses&>(synapses);
    fAllowBackPropagation_ = spSynapses.allowBackPropagation();
}

/*
 * AllIZHNeurons.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllIZHNeurons.h"
#include "AllNeuronsDeviceFuncs.h"

#include "Book.h"

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllIZHNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 */
void AllIZHNeurons::allocDeviceStruct( AllIZHNeuronsDeviceProperties &allNeurons ) {
	int count = Simulator::getInstance().getTotalNeurons();

	AllIFNeurons::allocDeviceStruct( allNeurons ); 
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Aconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Bconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Dconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.u_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C3_, count * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllIZHNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 */
void AllIZHNeurons::deleteDeviceStruct( AllIZHNeuronsDeviceProperties& allNeurons ) {
	HANDLE_ERROR( hipFree( allNeurons.Aconst_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Bconst_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Cconst_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Dconst_ ) );
	HANDLE_ERROR( hipFree( allNeurons.u_ ) );
	HANDLE_ERROR( hipFree( allNeurons.C3_ ) );

	AllIFNeurons::deleteDeviceStruct( allNeurons );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllIZHNeurons::copyNeuronHostToDevice( void* allNeuronsDevice ) { 
	AllIZHNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 */
void AllIZHNeurons::copyHostToDevice( AllIZHNeuronsDeviceProperties& allNeurons ) { 
	int count = Simulator::getInstance().getTotalNeurons();

	AllIFNeurons::copyHostToDevice( allNeurons );

	HANDLE_ERROR( hipMemcpy ( allNeurons.Aconst_, Aconst_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Bconst_, Bconst_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cconst_, Cconst_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Dconst_, Dconst_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.u_, u_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C3_, C3_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllIZHNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 */
void AllIZHNeurons::copyDeviceToHost( AllIZHNeuronsDeviceProperties& allNeurons ) {
	int count = Simulator::getInstance().getTotalNeurons();

	AllIFNeurons::copyDeviceToHost( allNeurons );

	HANDLE_ERROR( hipMemcpy ( Aconst_, allNeurons.Aconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Bconst_, allNeurons.Bconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cconst_, allNeurons.Cconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Dconst_, allNeurons.Dconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( u_, allNeurons.u_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C3_, allNeurons.C3_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice ) {
        AllIZHNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice )
{
        AllIZHNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 */
void AllIZHNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice )
{
        AllIZHNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons );
}

/*
 *  Notify outgoing synapses if neuron has fired.
 *
 */
void AllIZHNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, float* randNoise, SynapseIndexMap* synapseIndexMapDevice)
{
    int neuron_count = Simulator::getInstance().getTotalNeurons();
    int maxSpikes = (int)((Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate()));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceIZHNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, Simulator::getInstance().getMaxSynapsesPerNeuron(), maxSpikes, Simulator::getInstance().getDeltaT(), g_simulationStep, randNoise, (AllIZHNeuronsDeviceProperties *)allNeuronsDevice, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice, synapseIndexMapDevice, fAllowBackPropagation_ );
}

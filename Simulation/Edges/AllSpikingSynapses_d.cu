/*
 * AllSpikingSynapses.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice ) {
        allocSynapseDeviceStruct( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        allocDeviceStruct( allSynapses, numNeurons, maxSynapsesPerNeuron );

        HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocDeviceStruct( AllSpikingSynapsesDeviceProperties &allSynapses, int numNeurons, int maxSynapsesPerNeuron ) {
        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.sourceNeuronIndex_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.destNeuronIndex_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.W_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.type_, maxTotalSynapses * sizeof( synapseType ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.psr_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.inUse_, maxTotalSynapses * sizeof( bool ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapseCounts_, numNeurons * sizeof( BGSIZE ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.decay_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tau_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.totalDelay_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueue_, maxTotalSynapses * sizeof( uint32_t ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayIndex_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueueLength_, maxTotalSynapses * sizeof( int ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        deleteDeviceStruct( allSynapses );

        HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteDeviceStruct( AllSpikingSynapsesDeviceProperties& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.sourceNeuronIndex_ ) );
        HANDLE_ERROR( hipFree( allSynapses.destNeuronIndex_ ) );
        HANDLE_ERROR( hipFree( allSynapses.W_ ) );
        HANDLE_ERROR( hipFree( allSynapses.type_ ) );
        HANDLE_ERROR( hipFree( allSynapses.psr_ ) );
        HANDLE_ERROR( hipFree( allSynapses.inUse_ ) );
        HANDLE_ERROR( hipFree( allSynapses.synapseCounts_ ) );
        HANDLE_ERROR( hipFree( allSynapses.decay_ ) );
        HANDLE_ERROR( hipFree( allSynapses.tau_ ) );
        HANDLE_ERROR( hipFree( allSynapses.totalDelay_ ) );
        HANDLE_ERROR( hipFree( allSynapses.delayQueue_ ) );
        HANDLE_ERROR( hipFree( allSynapses.delayIndex_ ) );
        HANDLE_ERROR( hipFree( allSynapses.delayQueueLength_ ) );

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.countNeurons_ = 0;
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice ) { // copy everything necessary
        copySynapseHostToDevice( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyHostToDevice( allSynapsesDevice, allSynapses, numNeurons, maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyHostToDevice( void* allSynapsesDevice, AllSpikingSynapsesDeviceProperties& allSynapses, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;

        allSynapses.maxSynapsesPerNeuron_ = maxSynapsesPerNeuron_;
        allSynapses.totalSynapseCount_ = totalSynapseCount_;
        allSynapses.countNeurons_ = countNeurons_;
        HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( allSynapses.sourceNeuronIndex_, sourceNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.destNeuronIndex_, destNeuronIndex_,
                maxTotalSynapses * sizeof( int ),  hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.W_, W_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.type_, type_,
                maxTotalSynapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.psr_, psr_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.inUse_, inUse_,
                maxTotalSynapses * sizeof( bool ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.synapseCounts_, synapseCounts_,
                        numNeurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.decay_, decay_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.tau_, tau_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.totalDelay_, totalDelay_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayQueue_, delayQueue_,
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayIndex_, delayIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayQueueLength_, delayQueueLength_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::copySynapseDeviceToHost( void* allSynapsesDevice ) {
        // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyDeviceToHost( allSynapses );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyDeviceToHost( AllSpikingSynapsesDeviceProperties& allSynapses ) {
        int numNeurons = Simulator::getInstance().getTotalNeurons();
        BGSIZE maxTotalSynapses = Simulator::getInstance().getMaxSynapsesPerNeuron() * numNeurons;

        HANDLE_ERROR( hipMemcpy ( synapseCounts_, allSynapses.synapseCounts_,
                numNeurons * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuron_ = allSynapses.maxSynapsesPerNeuron_;
        totalSynapseCount_ = allSynapses.totalSynapseCount_;
        countNeurons_ = allSynapses.countNeurons_;

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex_, allSynapses.sourceNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndex_, allSynapses.destNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( W_, allSynapses.W_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( type_, allSynapses.type_,
                maxTotalSynapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psr_, allSynapses.psr_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUse_, allSynapses.inUse_,
                maxTotalSynapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decay_, allSynapses.decay_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tau_, allSynapses.tau_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( totalDelay_, allSynapses.totalDelay_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueue_, allSynapses.delayQueue_,
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIndex_, allSynapses.delayIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueueLength_, allSynapses.delayQueueLength_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
}

/*
 *  Get synapse_counts in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::copyDeviceSynapseCountsToHost( void* allSynapsesDevice )
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        int neuronCount = Simulator::getInstance().getTotalNeurons();

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCounts_, allSynapses.synapseCounts_, neuronCount * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.countNeurons_ = 0;
}

/* 
 *  Get summationCoord and in_use in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct
 *                             on device memory.
 */
void AllSpikingSynapses::copyDeviceSynapseSumIdxToHost(void* allSynapsesDevice )
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        BGSIZE maxTotalSynapses = Simulator::getInstance().getMaxSynapsesPerNeuron() * Simulator::getInstance().getTotalNeurons();

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex_, allSynapses.sourceNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUse_, allSynapses.inUse_,
                maxTotalSynapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
       
        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.countNeurons_ = 0;
}

/*
 *  Set some parameters used for advanceSynapsesDevice.
 */
void AllSpikingSynapses::setAdvanceSynapsesDeviceParams()
{
    setSynapseClassID();
}

/**
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSpikingSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSpikingSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses) ) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      GPU address of the AllSynapsesDeviceProperties struct
 *                                 on device memory.
 *  @param  allNeuronsDevice       GPU address of the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  GPU address of the SynapseIndexMap on device memory.
 */
void AllSpikingSynapses::advanceSynapses(void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice )
{
    if (totalSynapseCount_ == 0)
        return;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( totalSynapseCount_ + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceSpikingSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( totalSynapseCount_, (SynapseIndexMap*) synapseIndexMapDevice, g_simulationStep, Simulator::getInstance().getDeltaT(), (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice );
}

/*
 *  Prints GPU SynapsesProps data.
 *   
 *  @param  allSynapsesDeviceProps   GPU address of the corresponding SynapsesDeviceProperties struct on device memory.
 */
void AllSpikingSynapses::printGPUSynapsesProps( void* allSynapsesDeviceProps ) const
{
    AllSpikingSynapsesDeviceProperties allSynapsesProps;

    //allocate print out data members
    BGSIZE size = maxSynapsesPerNeuron_ * countNeurons_;
    if (size != 0) {
        BGSIZE *synapseCountsPrint = new BGSIZE[countNeurons_];
        BGSIZE maxSynapsesPerNeuronPrint;
        BGSIZE totalSynapseCountPrint;
        int countNeuronsPrint;
        int *sourceNeuronIndexPrint = new int[size];
        int *destNeuronIndexPrint = new int[size];
        BGFLOAT *WPrint = new BGFLOAT[size];

        synapseType *typePrint = new synapseType[size];
        BGFLOAT *psrPrint = new BGFLOAT[size];
        bool *inUsePrint = new bool[size];

        for (BGSIZE i = 0; i < size; i++) {
            inUsePrint[i] = false;
        }

        for (int i = 0; i < countNeurons_; i++) {
            synapseCountsPrint[i] = 0;
        }

        BGFLOAT *decayPrint = new BGFLOAT[size];
        int *totalDelayPrint = new int[size];
        BGFLOAT *tauPrint = new BGFLOAT[size];


        // copy everything
        HANDLE_ERROR( hipMemcpy ( &allSynapsesProps, allSynapsesDeviceProps, sizeof( AllSpikingSynapsesDeviceProperties), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCountsPrint, allSynapsesProps.synapseCounts_, countNeurons_ * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuronPrint = allSynapsesProps.maxSynapsesPerNeuron_;
        totalSynapseCountPrint = allSynapsesProps.totalSynapseCount_;
        countNeuronsPrint = allSynapsesProps.countNeurons_;

        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndexPrint, allSynapsesProps.sourceNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndexPrint, allSynapsesProps.destNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WPrint, allSynapsesProps.W_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( typePrint, allSynapsesProps.type_, size * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psrPrint, allSynapsesProps.psr_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUsePrint, allSynapsesProps.inUse_, size * sizeof( bool ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decayPrint, allSynapsesProps.decay_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauPrint, allSynapsesProps.tau_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( totalDelayPrint, allSynapsesProps.totalDelay_, size * sizeof( int ), hipMemcpyDeviceToHost ) );


        for(int i = 0; i < maxSynapsesPerNeuron_ * countNeurons_; i++) {
            if (WPrint[i] != 0.0) {
                cout << "GPU W[" << i << "] = " << WPrint[i];
                cout << " GPU sourNeuron: " << sourceNeuronIndexPrint[i];
                cout << " GPU desNeuron: " << destNeuronIndexPrint[i];
                cout << " GPU type: " << typePrint[i];
                cout << " GPU psr: " << psrPrint[i];
                cout << " GPU in_use:" << inUsePrint[i];

                cout << " GPU decay: " << decayPrint[i];
                cout << " GPU tau: " << tauPrint[i];
                cout << " GPU total_delay: " << totalDelayPrint[i] << endl;;
            }
        }

        for (int i = 0; i < countNeurons_; i++) {
            cout << "GPU synapse_counts:" << "neuron[" << i  << "]" << synapseCountsPrint[i] << endl;
        }

        cout << "GPU totalSynapseCount:" << totalSynapseCountPrint << endl;
        cout << "GPU maxSynapsesPerNeuron:" << maxSynapsesPerNeuronPrint << endl;
        cout << "GPU countNeurons_:" << countNeuronsPrint << endl;


        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllDSSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        delete[] destNeuronIndexPrint;
        delete[] WPrint;
        delete[] sourceNeuronIndexPrint;
        delete[] psrPrint;
        delete[] typePrint;
        delete[] inUsePrint;
        delete[] synapseCountsPrint;
        destNeuronIndexPrint = NULL;
        WPrint = NULL;
        sourceNeuronIndexPrint = NULL;
        psrPrint = NULL;
        typePrint = NULL;
        inUsePrint = NULL;
        synapseCountsPrint = NULL;

        delete[] decayPrint;
        delete[] totalDelayPrint;
        delete[] tauPrint;
        decayPrint = NULL;
        totalDelayPrint = NULL;
        tauPrint = NULL;
    }
}



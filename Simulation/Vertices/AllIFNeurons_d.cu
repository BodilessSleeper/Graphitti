/*
 * AllIFNeurons_d.cu
 *
 */

#include "AllIFNeurons.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice ) {
	AllIFNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::allocDeviceStruct( AllIFNeuronsDeviceProperties &allNeurons ) {
	int count = Simulator::getInstance().getTotalNeurons();
	int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C1_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C2_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.I0_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Iinject_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Inoise_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Isyn_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Rm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Tau_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Trefract_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vinit_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vreset_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vrest_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vthresh_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.hasFired_, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.numStepsInRefractoryPeriod_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCount_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCountOffset_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.summationMap_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeHistory_, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &pSpikeHistory[i], maxSpikes * sizeof( uint64_t ) ) );
	}
	HANDLE_ERROR( hipMemcpy ( allNeurons.spikeHistory_, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address
	summationMap_ = allNeurons.summationMap_;
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice ) {
	AllIFNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::deleteDeviceStruct( AllIFNeuronsDeviceProperties& allNeurons ) {
	int count = Simulator::getInstance().getTotalNeurons();

	uint64_t* pSpikeHistory[count];
	HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spikeHistory_,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipFree( pSpikeHistory[i] ) );
	}

	HANDLE_ERROR( hipFree( allNeurons.C1_ ) );
	HANDLE_ERROR( hipFree( allNeurons.C2_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Cm_ ) );
	HANDLE_ERROR( hipFree( allNeurons.I0_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Iinject_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Inoise_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Isyn_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Rm_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Tau_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Trefract_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Vinit_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Vm_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Vreset_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Vrest_ ) );
	HANDLE_ERROR( hipFree( allNeurons.Vthresh_ ) );
	HANDLE_ERROR( hipFree( allNeurons.hasFired_ ) );
	HANDLE_ERROR( hipFree( allNeurons.numStepsInRefractoryPeriod_ ) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCount_ ) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCountOffset_ ) );
	HANDLE_ERROR( hipFree( allNeurons.summationMap_ ) );
	HANDLE_ERROR( hipFree( allNeurons.spikeHistory_ ) );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronHostToDevice( void* allNeuronsDevice ) { 
	AllIFNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::copyHostToDevice( AllIFNeuronsDeviceProperties& allNeurons ) { 
	int count = Simulator::getInstance().getTotalNeurons();

	HANDLE_ERROR( hipMemcpy ( allNeurons.C1_, C1_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C2_, C2_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cm_, Cm_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.I0_, I0_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Iinject_, Iinject_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Inoise_, Inoise_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Isyn_, Isyn_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Rm_, Rm_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Tau_, Tau_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Trefract_, Trefract_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vinit_, Vinit_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vm_, Vm_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vreset_, Vreset_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vrest_, Vrest_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vthresh_, Vthresh_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.hasFired_, hasFired_, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.numStepsInRefractoryPeriod_, numStepsInRefractoryPeriod_, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCount_, spikeCount_, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCountOffset_, spikeCountOffset_, count * sizeof( int ), hipMemcpyHostToDevice ) );

        int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spikeHistory_, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( pSpikeHistory[i], spikeHistory_[i], maxSpikes * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        }
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice ) {
	AllIFNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::copyDeviceToHost( AllIFNeuronsDeviceProperties& allNeurons ) {
	int count = Simulator::getInstance().getTotalNeurons();

	HANDLE_ERROR( hipMemcpy ( C1_, allNeurons.C1_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C2_, allNeurons.C2_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cm_, allNeurons.Cm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( I0_, allNeurons.I0_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Iinject_, allNeurons.Iinject_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Inoise_, allNeurons.Inoise_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Isyn_, allNeurons.Isyn_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Rm_, allNeurons.Rm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Tau_, allNeurons.Tau_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Trefract_, allNeurons.Trefract_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vinit_, allNeurons.Vinit_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vm_, allNeurons.Vm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vreset_, allNeurons.Vreset_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vrest_, allNeurons.Vrest_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vthresh_, allNeurons.Vthresh_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( hasFired_, allNeurons.hasFired_, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( numStepsInRefractoryPeriod_, allNeurons.numStepsInRefractoryPeriod_, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCount_, allNeurons.spikeCount_, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCountOffset_, allNeurons.spikeCountOffset_, count * sizeof( int ), hipMemcpyDeviceToHost ) );

        int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spikeHistory_, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( spikeHistory_[i], pSpikeHistory[i], maxSpikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice ) 
{        
        AllIFNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );        
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice )
{
        AllIFNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice )
{
        AllIFNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons );
}


/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       GPU address of the AllIFNeuronsDeviceProperties struct 
 *                                 on device memory.
 *  @param  allSynapsesDevice      GPU address of the allSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  GPU address of the SynapseIndexMap on device memory.
 */
void AllIFNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, float* randNoise, SynapseIndexMap* synapseIndexMapDevice )
{
}
